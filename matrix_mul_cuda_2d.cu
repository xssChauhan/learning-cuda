#include "hip/hip_runtime.h"
/*
 * Multiplying a 2D matrix using CUDA 
 */

#include<stdio.h>
#include<stdlib.h>
#include<assert.h>

#define BLOCK_SIZE 16

__global__ void gpu_matrix_mul( int *a, int *b, int *c, int m, int n, int k){
  int row = blockIdx.y + blockDim.y * threadIdx.y;
  int col = blockIdx.x + blockDim.x * threadIdx.x;
  int sum = 0;

  if(col < k && row < m){
    for(int i = 0; i < n; i++){
      sum += a[row*n + i] * b[i*k + col];
    }
    c[row * k + col] = sum;
  }
}

void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
  for (int i = 0; i < m; ++i) 
  {
    for (int j = 0; j < k; ++j) 
    {
      int tmp = 0.0;
      for (int h = 0; h < n; ++h) 
      {
        tmp += h_a[i * n + h] * h_b[h * k + j];
      }
      h_result[i * k + j] = tmp;
    }
  }
}

int main(){
  int m,n,k; // m=rows of 1st , n= cols of 1st and rows of 2nd , k = cols of 2nd
  srand(3333);

  m = 1024;
  n = 1024;
  k = 1024;

  //Allocate memory in host RAM, h_cc is used to store CPU results
  int *h_a, *h_b, *h_c , *h_cc;

  hipHostMalloc((void **) &h_a , sizeof(int)*m*n);
  hipHostMalloc((void **) &h_b , sizeof(int)*n*k);
  hipHostMalloc((void **) &h_c , sizeof(int)*m*k);
  hipHostMalloc((void **) &h_cc , sizeof(int)*m*k);

  //random initialie matrix A

  for(int i= 0; i < m; ++i){
    for(int j=0; j < n; ++j){
      h_a[i*n + j ] = rand() %1024;
    }
  }

  //Random intialize B
  for( int i = 0; i<n; ++i){
    for( int j = 0; j < n; j++){
      h_b[i*k + j] = rand()%1024;
    }
  }
  
  float gpu_elapsed_time_ms;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  
  //Allocate Memory on the device
  int *d_a, *d_b, *d_c;
  hipMalloc((void **) &d_a, sizeof(int)*m*n);
  hipMalloc((void **) &d_b, sizeof(int)*n*k);
  hipMalloc((void **) &d_c, sizeof(int)*m*k);
 
  //Copy matrix A and B from host to device memory
  hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

  unsigned int grid_rows = (m + BLOCK_SIZE -1 ) / BLOCK_SIZE;
  unsigned int grid_cols = (k + BLOCK_SIZE -1 )/ BLOCK_SIZE;

  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE);

  gpu_matrix_mul<<<dimGrid, dimBlock>>>(
        d_a, d_b, d_c, m, n, k 
    );
  hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  
  hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
  printf("Time Elapsed on matrix multiplication of %dx%d . %dx%d on GPU : %fms.\n\n", m,n,n,n,k, gpu_elapsed_time_ms);

  float cpu_elapsed_time_ms;

  hipEventRecord(start, 0);

  cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
  printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on CPU: %f ms.\n\n", m, n, n, k, cpu_elapsed_time_ms);

}
